#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<thrust/device_ptr.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>
#include "dmatrix.h"

DMCudaArray* dmCudaInitArray(const IOFile *file)
{
	// validate input arguments
	if( NULL==file )
	{
		throw std::string("invalid IOFile object");
	}

	DMCudaArray *array=NULL;

	try
	{
		array=new DMCudaArray;
		array->cols=file->getCols();
		array->rows=file->getRows();
		array->bytes=sizeof(float)*array->cols*array->rows;
		array->ordering=file->getMajorOrdering();

		cudaASSERT( hipMalloc(&array->pointer,array->bytes) );
		cudaASSERT( hipMemcpy(array->pointer,file->getDataPtr(),array->bytes,hipMemcpyHostToDevice) );
	}
	catch( const std::string& error )
	{
		if( NULL!=array )
		{
			dmCudaFree(array);
		}
		throw "cannot initialize the CUDA array ("+error+")";
	}

	return array;
}

__global__ void dmCudaInitIndexes(int *indexes, size_t numel)
{
	// compute index
	const int x=blockDim.x*blockIdx.x+threadIdx.x;

    if( numel>x )
    {
		// assign index value
		indexes[x]=x;
    }
}

DMCudaArray* dmCudaSortArray(const DMCudaArray *keys, size_t bsize)
{
	// validate input arguments
	if( NULL==keys || 1!=keys->cols )
	{
		throw std::string("invalid keys array");
	}

	DMCudaArray *indexes=NULL;

	try
	{
		indexes=new DMCudaArray;
		indexes->cols=1;
		indexes->rows=keys->rows;
		indexes->bytes=sizeof(int)*indexes->cols*indexes->rows;
		indexes->ordering=IOColMajor;

		cudaASSERT( hipMalloc(&indexes->pointer,indexes->bytes) );

		dim3 grid(1,1,1);
		dim3 threads(bsize,1,1);
		grid.x=(indexes->rows/bsize)+((indexes->rows%bsize)?1:0);

		dmCudaInitIndexes<<<grid,threads>>>((int*)indexes->pointer,indexes->rows);
		cudaASSERT( hipPeekAtLastError() );
		cudaASSERT( hipDeviceSynchronize() );

		thrust::device_ptr<float> t_keys((float*)keys->pointer);
		thrust::device_ptr<int> t_values((int*)indexes->pointer);
		thrust::sort_by_key(t_keys,t_keys+keys->rows,t_values);
	}
	catch( const std::string& error )
	{
		if( NULL!=indexes )
		{
			dmCudaFree(indexes);
		}
		throw "cannot initialize the indexes array ("+error+")";
	}

	return indexes;
}

DMCudaArray* dmCudaCountKeys(const DMCudaArray *keys)
{
	// validate input arguments
	if( NULL==keys || 1!=keys->cols )
	{
		throw std::string("invalid keys array");
	}

	// temporary copy keys to the host
	float *aux=new float[keys->rows];
	cudaASSERT( hipMemcpy(aux,keys->pointer,keys->bytes,hipMemcpyDeviceToHost) );

	// count unique keys and compute frequencies
	int i=0,j=1;
	std::vector<int> frequencies;
	for( ; keys->rows-1>i; i++,j++ )
	{
		if( (aux[i+1]-aux[i]) )
		{
			frequencies.push_back(j);
			j=0;
		}
	}
	frequencies.push_back(j);
/*
for( int i=0; frequencies.size()>i; i++ )
{
	std::cout << frequencies[i] << ", ";
}
std::cout << frequencies.size() << std::endl;
*/
	DMCudaArray *count=NULL;

	try
	{
		count=new DMCudaArray;
		count->cols=1;
		count->rows=frequencies.size();
		count->bytes=sizeof(int)*count->cols*count->rows;
		count->ordering=IOColMajor;

		cudaASSERT( hipMalloc(&count->pointer,count->bytes) );
		cudaASSERT( hipMemcpy(count->pointer,&frequencies[0],count->bytes,hipMemcpyHostToDevice) );
	}
	catch( const std::string& error )
	{
		if( NULL!=count )
		{
			dmCudaFree(count);
		}
		throw "cannot initialize keys frequencies array ("+error+")";
	}

	return count;
}

void dmCudaFree(DMCudaArray *array)
{
	if( NULL!=array )
	{
		if( NULL!=array->pointer )
		{
			// release the cuda array
			cudaASSERT( hipFree(array->pointer) );
		}

		// clear memory resources
		std::memset(array,sizeof(array),0);

		// release memory resources
		delete array;
	}
}
