#include<sstream>
#include<hip/hip_runtime.h>

// throw exception when a CUDA error occurs
void cudaAssert(int code, const char *file, const char *fn, int line)
{
	if( hipSuccess!=code ) 
	{
		std::stringstream ss;
		ss << hipGetErrorString((hipError_t)code) << " at " << file << ":" << line;
		throw ss.str();
	}
}
