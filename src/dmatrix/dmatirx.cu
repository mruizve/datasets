#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<iomanip>
#include "dmatrix.h"

__global__ void dmCudaDistancesColMajor(float *matrix, size_t mdim, const float *features, size_t fcols, size_t frows, const int *indexes, const int *count)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(i)*frows+(j)]
	#define M(i,j,k) matrix[3*((j)*mdim+(i))+k]
	if( mdim>x && mdim>y && x>=y )
	{
		// compute indexes offsets
		int xoffset=0;
		for( int i=0; x>i; i++ )
		{
			xoffset+=count[i];
		}

		int yoffset=0;
		for( int i=0; y>i; i++ )
		{
			yoffset+=count[i];
		}

		// distances initialization
		float d_mean=0.0f,d_max=0.0f,d_min=1e9;

		// compute distances between all images of the labels pair
		for( int i=0; count[x]>i; i++)
		{
			for( int j=0; count[y]>j; j++)
			{
				// compute indexes
				int ii=indexes[xoffset+i];
				int jj=indexes[yoffset+j];

				// compute features distances
				float d=0.0f;
				for( int k=0; fcols>k; k++ )
				{
					d+=(F(k,ii)-F(k,jj))*(F(k,ii)-F(k,jj));
				}
				d=sqrtf(d);

				// accumulate mean
				d_mean+=d;

				// get maximum distance
				d_max=fmaxf(d,d_max);

				// get minimum distance
				d_min=fminf(d,d_min);
			}
		}

		// compute mean
		d_mean=d_mean/(count[x]*count[y]);

		// update matrix values
		M(x,y,0)=d_mean;
		M(x,y,1)=d_max;
		M(x,y,2)=d_min;

		M(y,x,0)=d_mean;
		M(y,x,1)=d_max;
		M(y,x,2)=d_min;
	}
	#undef M
    #undef F
}

__global__ void dmCudaDistancesRowMajor(float *matrix, size_t mdim, const float *features, size_t fcols, size_t frows, const int *indexes, const int *count)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(j)*fcols+(i)]
	#define M(i,j,k) matrix[3*((j)*mdim+(i))+k]
	if( mdim>x && mdim>y && x>=y )
	{
		// compute indexes offsets
		int xoffset=0;
		for( int i=0; x>i; i++ )
		{
			xoffset+=count[i];
		}

		int yoffset=0;
		for( int i=0; y>i; i++ )
		{
			yoffset+=count[i];
		}

		// distances initialization
		float d_mean=0.0f,d_max=0.0f,d_min=1e9;

		// compute distances between all images of the labels pair
		for( int i=0; count[x]>i; i++)
		{
			for( int j=0; count[y]>j; j++)
			{
				// compute indexes
				int ii=indexes[xoffset+i];
				int jj=indexes[yoffset+j];

				// compute features distances
				float d=0.0f;
				for( int k=0; fcols>k; k++ )
				{
					d+=(F(k,ii)-F(k,jj))*(F(k,ii)-F(k,jj));
				}
				d=sqrtf(d);

				// accumulate mean
				d_mean+=d;

				// get maximum distance
				d_max=fmaxf(d,d_max);

				// get minimum distance
				d_min=fminf(d,d_min);
			}
		}

		// compute mean
		d_mean=d_mean/(count[x]*count[y]);

		// update matrix values
		M(x,y,0)=d_mean;
		M(x,y,1)=d_max;
		M(x,y,2)=d_min;

		M(y,x,0)=d_mean;
		M(y,x,1)=d_max;
		M(y,x,2)=d_min;
	}
	#undef M
    #undef F
}

__global__ void dmCudaDMColMajor
(
	float *distances, const float *features, size_t fcols, size_t frows, const int *indexes, size_t xoffset, size_t yoffset, size_t xdim, size_t ydim
)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(i)*frows+(j)]
	#define D(i,j) distances[(j)*xdim+(i)]
	if( xdim>x && ydim>y )
	{
		// compute features indexes
		int xx=indexes[xoffset+x];
		int yy=indexes[yoffset+y];
		
		// compute features distances
		float d=0.0f;
		for( int k=0; fcols>k; k++ )
		{
			d+=(F(k,xx)-F(k,yy))*(F(k,xx)-F(k,yy));
		}

		D(x,y)=sqrtf(d);
	}
	#undef D
	#undef F
}

__global__ void dmCudaDMRowMajor
(
	float *distances, const float *features, size_t fcols, size_t frows, const int *indexes, size_t xoffset, size_t yoffset, size_t xdim, size_t ydim
)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(j)*fcols+(i)]
	#define D(i,j) distances[(j)*xdim+(i)]
	if( xdim>x && ydim>y )
	{
		// compute features indexes
		int xx=indexes[xoffset+x];
		int yy=indexes[yoffset+y];
		
		// compute features distances
		float d=0.0f;
		for( int k=0; fcols>k; k++ )
		{
			d+=(F(k,xx)-F(k,yy))*(F(k,xx)-F(k,yy));
		}

		D(x,y)=sqrtf(d);
	}
	#undef D
	#undef F
}

//DMCudaArray* dmCudaDistanceMatrix(const DMCudaArray *features, const DMCudaArray *indexes, const DMCudaArray *count, size_t bsize)
cv::Mat dmCudaDistanceMatrix(const DMCudaArray *features, const DMCudaArray *indexes, const std::vector<int> offsets, size_t bsize)
{
	// validate input arguments
	if( NULL==features )
	{
		throw std::string("invalid features array");
	}

	if( NULL==indexes || 1!=indexes->cols )
	{
		throw std::string("invalid indexes array");
	}

	if( features->rows!=indexes->rows )
	{
		throw std::string("features and indexes arrays should have the same number of elements");
	}
	
	// if( NULL==count || 1!=count->cols || indexes->rows<count->rows )
	// {
		// throw std::string("invalid frequencies array");
	// }
	if( indexes->rows<(offsets.size()-1) )
	{
		throw std::string("invalid offsets array");
	}
//	DMCudaArray *matrix=NULL;

	try
	{
/*
		// array initialization
		matrix=new DMCudaArray;
		matrix->cols=count->rows;
		matrix->rows=count->rows;
		matrix->bytes=3*sizeof(float)*matrix->cols*matrix->rows;
		matrix->ordering=IORowMajor;

		cudaASSERT( hipMalloc(&matrix->pointer,matrix->bytes) );

		dim3 grid(1,1,1);
		dim3 threads(bsize,bsize,1);
		grid.x=(matrix->cols/bsize)+((matrix->cols%bsize)?1:0);
		grid.y=(matrix->rows/bsize)+((matrix->rows%bsize)?1:0);

		if( IOColMajor==features->ordering )
		{
			dmCudaDistancesColMajor<<<grid,threads>>>(
				(float*)matrix->pointer,matrix->cols,
				(float*)features->pointer,features->cols,features->rows,
				(int*)indexes->pointer,(int*)count->pointer);
		}
		else
		{
			dmCudaDistancesRowMajor<<<grid,threads>>>(
				(float*)matrix->pointer,matrix->cols,
				(float*)features->pointer,features->cols,features->rows,
				(int*)indexes->pointer,(int*)count->pointer);
		}
*/
		cv::Mat matrix(offsets.size()-1,offsets.size()-1,CV_32FC3);

		for( size_t i=0; (offsets.size()-1)>i; i++ )
		{
			size_t xdim=offsets[i+1]-offsets[i];

			for( size_t j=i; (offsets.size()-1)>j; j++ )
			{
				size_t ydim=offsets[j+1]-offsets[j];

				float *d_distances;
				size_t bytes=sizeof(float)*xdim*ydim;
				cudaASSERT( hipMalloc((void**)&d_distances,bytes) );

				dim3 grid(1,1,1);
				dim3 threads(bsize,bsize,1);
				grid.x=(xdim/bsize)+((xdim%bsize)?1:0);
				grid.y=(ydim/bsize)+((ydim%bsize)?1:0);

				// computed distances between all features of the labels pair
				if( IOColMajor==features->ordering )
				{
					dmCudaDMColMajor<<<grid,threads>>>(
						d_distances,
						(float*)features->pointer,features->cols,features->rows,
						(int*)indexes->pointer,
						offsets[i],offsets[j],xdim,ydim);
				}
				else
				{
					dmCudaDMRowMajor<<<grid,threads>>>(
						d_distances,
						(float*)features->pointer,features->cols,features->rows,
						(int*)indexes->pointer,
						offsets[i],offsets[j],xdim,ydim);
				}

				cudaASSERT( hipPeekAtLastError() );

				// retrieve distances
				std::vector<float> h_distances(xdim*ydim);
				cudaASSERT( hipMemcpy(&h_distances[0],d_distances,bytes,hipMemcpyDeviceToHost) );

				// free memory resources
				cudaASSERT( hipFree(d_distances) );

				// reduce distances
				float d_mean=0.0f,d_max=0.0f,d_min=1e9;
				for( size_t k=0; h_distances.size()>k; k++ )
				{
					d_mean+=h_distances.at(k);
					d_max=std::max(d_max,h_distances.at(k));
					d_min=std::min(d_min,h_distances.at(k));
				}
				d_mean=d_mean/h_distances.size();

				// store distances
				matrix.at<cv::Vec3f>(i,j)[0]=d_mean;
				matrix.at<cv::Vec3f>(i,j)[1]=d_max;
				matrix.at<cv::Vec3f>(i,j)[2]=d_min;

				matrix.at<cv::Vec3f>(j,i)=matrix.at<cv::Vec3f>(i,j);
			}

			std::cout
				<< std::setw(5) << std::setfill('0') << i << "/"
				<< std::setw(5) << std::setfill('0') << offsets.size()-1 <<'\r'
				<< std::flush;
		}
		std::cout << std::endl;

		return matrix;
/*
		cudaASSERT( hipPeekAtLastError() );
		cudaASSERT( hipDeviceSynchronize() );
*/
	}
	catch( const std::string& error )
	{
/*
		if( NULL!=matrix )
		{
			dmCudaFree(matrix);
		}
*/
		throw "cannot generate the distance matrix array ("+error+")";
	}

//	return matrix;
	return cv::Mat();
}
