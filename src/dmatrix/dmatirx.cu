#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include "dmatrix.h"

__global__ void dmCudaDistancesColMajor(float *matrix, size_t mdim, const float *features, size_t fcols, size_t frows, const int *indexes, const int *count)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(i)*frows+(j)]
	#define M(i,j,k) matrix[3*((j)*mdim+(i))+k]
	if( mdim>x && mdim>y && x>=y )
	{
		// compute indexes offsets
		int xoffset=0;
		for( int i=0; x>i; i++ )
		{
			xoffset+=count[i];
		}

		int yoffset=0;
		for( int i=0; y>i; i++ )
		{
			yoffset+=count[i];
		}

		// distances initialization
		float d_mean=0.0f,d_max=0.0f,d_min=1e9;

		// compute distances between all images of the labels pair
		for( int i=0; count[x]>i; i++)
		{
			for( int j=0; count[y]>j; j++)
			{
				// compute indexes
				int ii=indexes[xoffset+i];
				int jj=indexes[yoffset+j];

				// compute features distances
				float d=0.0f;
				for( int k=0; fcols>k; k++ )
				{
					d+=(F(k,ii)-F(k,jj))*(F(k,ii)-F(k,jj));
				}
				d=sqrtf(d);

				// accumulate mean
				d_mean+=d;

				// get maximum distance
				d_max=fmaxf(d,d_max);

				// get minimum distance
				d_min=fminf(d,d_min);
			}
		}

		// compute mean
		d_mean=d_mean/(count[x]*count[y]);

		// update matrix values
		M(x,y,0)=d_mean;
		M(x,y,1)=d_max;
		M(x,y,2)=d_min;

		M(y,x,0)=d_mean;
		M(y,x,1)=d_max;
		M(y,x,2)=d_min;
	}
	#undef M
    #undef F
}

__global__ void dmCudaDistancesRowMajor(float *matrix, size_t mdim, const float *features, size_t fcols, size_t frows, const int *indexes, const int *count)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(j)*fcols+(i)]
	#define M(i,j,k) matrix[3*((j)*mdim+(i))+k]
	if( mdim>x && mdim>y && x>=y )
	{
		// compute indexes offsets
		int xoffset=0;
		for( int i=0; x>i; i++ )
		{
			xoffset+=count[i];
		}

		int yoffset=0;
		for( int i=0; y>i; i++ )
		{
			yoffset+=count[i];
		}

		// distances initialization
		float d_mean=0.0f,d_max=0.0f,d_min=1e9;

		// compute distances between all images of the labels pair
		for( int i=0; count[x]>i; i++)
		{
			for( int j=0; count[y]>j; j++)
			{
				// compute indexes
				int ii=indexes[xoffset+i];
				int jj=indexes[yoffset+j];

				// compute features distances
				float d=0.0f;
				for( int k=0; fcols>k; k++ )
				{
					d+=(F(k,ii)-F(k,jj))*(F(k,ii)-F(k,jj));
				}
				d=sqrtf(d);

				// accumulate mean
				d_mean+=d;

				// get maximum distance
				d_max=fmaxf(d,d_max);

				// get minimum distance
				d_min=fminf(d,d_min);
			}
		}

		// compute mean
		d_mean=d_mean/(count[x]*count[y]);

		// update matrix values
		M(x,y,0)=d_mean;
		M(x,y,1)=d_max;
		M(x,y,2)=d_min;

		M(y,x,0)=d_mean;
		M(y,x,1)=d_max;
		M(y,x,2)=d_min;
	}
	#undef M
    #undef F
}

DMCudaArray* dmCudaDistanceMatrix(const DMCudaArray *features, const DMCudaArray *indexes, const DMCudaArray *count, size_t bsize)
{
	// validate input arguments
	if( NULL==features )
	{
		throw std::string("invalid features array");
	}

	if( NULL==indexes || 1!=indexes->cols )
	{
		throw std::string("invalid indexes array");
	}

	if( features->rows!=indexes->rows )
	{
		throw std::string("features and indexes arrays should have the same number of elements");
	}
	
	if( NULL==count || 1!=count->cols || indexes->rows<count->rows )
	{
		throw std::string("invalid frequencies array");
	}

	DMCudaArray *matrix=NULL;

	try
	{
		// array initialization
		matrix=new DMCudaArray;
		matrix->cols=count->rows;
		matrix->rows=count->rows;
		matrix->bytes=3*sizeof(float)*matrix->cols*matrix->rows;
		matrix->ordering=IORowMajor;

		cudaASSERT( hipMalloc(&matrix->pointer,matrix->bytes) );

		dim3 grid(1,1,1);
		dim3 threads(bsize,bsize,1);
		grid.x=(matrix->cols/bsize)+((matrix->cols%bsize)?1:0);
		grid.y=(matrix->rows/bsize)+((matrix->rows%bsize)?1:0);

		if( IOColMajor==features->ordering )
		{
			dmCudaDistancesColMajor<<<grid,threads>>>(
				(float*)matrix->pointer,matrix->cols,
				(float*)features->pointer,features->cols,features->rows,
				(int*)indexes->pointer,(int*)count->pointer);
		}
		else
		{
			dmCudaDistancesRowMajor<<<grid,threads>>>(
				(float*)matrix->pointer,matrix->cols,
				(float*)features->pointer,features->cols,features->rows,
				(int*)indexes->pointer,(int*)count->pointer);
		}

		cudaASSERT( hipPeekAtLastError() );
		cudaASSERT( hipDeviceSynchronize() );
	}
	catch( const std::string& error )
	{
		if( NULL!=matrix )
		{
			dmCudaFree(matrix);
		}
		throw "cannot generate the distance matrix array ("+error+")";
	}

	return matrix;
}
