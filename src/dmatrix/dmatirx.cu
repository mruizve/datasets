#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<iomanip>
#include "dmatrix.h"

__global__ void dmCudaDistancesColMajor
(
	float *distances, const float *features, size_t fcols, size_t frows, const int *indexes, size_t xoffset, size_t yoffset, size_t xdim, size_t ydim
)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(i)*frows+(j)]
	#define D(i,j) distances[(j)*xdim+(i)]
	if( xdim>x && ydim>y )
	{
		// compute features indexes
		int xx=indexes[xoffset+x];
		int yy=indexes[yoffset+y];
		
		// compute features distances
		float d=0.0f;
		for( int k=0; fcols>k; k++ )
		{
			d+=(F(k,xx)-F(k,yy))*(F(k,xx)-F(k,yy));
		}

		D(x,y)=sqrtf(d);
	}
	#undef D
	#undef F
}

__global__ void dmCudaDistancesRowMajor
(
	float *distances, const float *features, size_t fcols, size_t frows, const int *indexes, size_t xoffset, size_t yoffset, size_t xdim, size_t ydim
)
{
	// compute pixel coordinates
	const int x=blockDim.x*blockIdx.x+threadIdx.x;
	const int y=blockDim.y*blockIdx.y+threadIdx.y;

	#define F(i,j) features[(j)*fcols+(i)]
	#define D(i,j) distances[(j)*xdim+(i)]
	if( xdim>x && ydim>y )
	{
		// compute features indexes
		int xx=indexes[xoffset+x];
		int yy=indexes[yoffset+y];
		
		// compute features distances
		float d=0.0f;
		for( int k=0; fcols>k; k++ )
		{
			d+=(F(k,xx)-F(k,yy))*(F(k,xx)-F(k,yy));
		}

		D(x,y)=sqrtf(d);
	}
	#undef D
	#undef F
}

cv::Mat dmCudaDistanceMatrix(const DMCudaArray *features, const DMCudaArray *indexes, const std::vector<int> offsets, size_t bsize)
{
	// validate input arguments
	if( NULL==features )
	{
		throw std::string("invalid features array");
	}

	if( NULL==indexes || 1!=indexes->cols )
	{
		throw std::string("invalid indexes array");
	}

	if( features->rows!=indexes->rows )
	{
		throw std::string("features and indexes arrays should have the same number of elements");
	}
	
	if( indexes->rows<(offsets.size()-1) )
	{
		throw std::string("invalid offsets array");
	}

	try
	{
		// distance matrix initialization
		cv::Mat matrix(offsets.size()-1,offsets.size()-1,CV_32FC4);
		float *raw=(float*)matrix.data;

		// computed distances between all features of the labels pair
		// (remember that count[i]=offsets[i+1]-offsets[i])
		std::cout << "processing distance matrix's rows" << std::endl;

		for( size_t i=0; (offsets.size()-1)>i; i++ )
		{
			size_t xdim=offsets[i+1]-offsets[i];

			for( size_t j=i; (offsets.size()-1)>j; j++ )
			{
				size_t ydim=offsets[j+1]-offsets[j];

				float *d_distances;
				size_t bytes=sizeof(float)*xdim*ydim;
				cudaASSERT( hipMalloc((void**)&d_distances,bytes) );

				dim3 grid(1,1,1);
				dim3 threads(bsize,bsize,1);
				grid.x=(xdim/bsize)+((xdim%bsize)?1:0);
				grid.y=(ydim/bsize)+((ydim%bsize)?1:0);

				if( IOColMajor==features->ordering )
				{
					dmCudaDistancesColMajor<<<grid,threads>>>(
						d_distances,
						(float*)features->pointer,features->cols,features->rows,
						(int*)indexes->pointer,
						offsets[i],offsets[j],xdim,ydim);
				}
				else
				{
					dmCudaDistancesRowMajor<<<grid,threads>>>(
						d_distances,
						(float*)features->pointer,features->cols,features->rows,
						(int*)indexes->pointer,
						offsets[i],offsets[j],xdim,ydim);
				}

				cudaASSERT( hipPeekAtLastError() );

				// retrieve distances
				std::vector<float> h_distances(xdim*ydim);
				cudaASSERT( hipMemcpy(&h_distances[0],d_distances,bytes,hipMemcpyDeviceToHost) );

				// free memory resources
				cudaASSERT( hipFree(d_distances) );

				// compute distances statistics
				float d_mean=0.0f,d_var=0.0f,d_max=0.0f,d_min=1e9;
				for( size_t k=0; h_distances.size()>k; k++ )
				{
					d_mean+=h_distances.at(k);
					d_max=std::max(d_max,h_distances.at(k));
					d_min=std::min(d_min,h_distances.at(k));
				}
				d_mean/=h_distances.size();
				for( size_t k=0; h_distances.size()>k; k++ )
				{
					d_var+=(h_distances.at(k)-d_mean)*(h_distances.at(k)-d_mean);
				}
				d_var/=(h_distances.size()-1);

				// store statistics
				raw[3*(i*matrix.cols+j)+0]=d_mean;
				raw[3*(i*matrix.cols+j)+1]=d_var;
				raw[3*(i*matrix.cols+j)+2]=d_max;
				raw[3*(i*matrix.cols+j)+3]=d_min;

				raw[3*(j*matrix.cols+i)+0]=d_mean;
				raw[3*(j*matrix.cols+i)+1]=d_var;
				raw[3*(j*matrix.cols+i)+2]=d_max;
				raw[3*(j*matrix.cols+i)+3]=d_min;
			}

			std::cout
				<< std::setw(5) << std::setfill('0') << i+1 << "/"
				<< std::setw(5) << std::setfill('0') << offsets.size()-1 <<'\r'
				<< std::flush;
		}
		std::cout << std::endl;

		return matrix;
	}
	catch( const std::string& error )
	{
		throw "cannot generate the distance matrix array ("+error+")";
	}
}
